#include "hip/hip_runtime.h"
#include <string.h>
#include <algorithm>
#include <cstdint>
#include <memory>
#include <mutex>
#include <vector>
// #include <boost/thread/mutex.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <gnuradio/cudabuffer.hpp>


// typedef boost::unique_lock<boost::mutex> scoped_lock;

namespace gr {
cuda_buffer::cuda_buffer(size_t num_items, size_t item_size, cuda_buffer_type type)
    : _num_items(num_items),
      _item_size(item_size),
      _buffer_type(type),
      _buf_size(_num_items * _item_size),
      _read_index(0),
      _write_index(0),
      _type(type)
{
    _host_buffer.resize(_buf_size * 2); // double circular buffer
    hipMalloc(
        &_device_buffer,
        _buf_size *
            2); // double circular buffer - should do something more intelligent here
    set_type("cuda_buffer_" + std::to_string((int)_type));
}
cuda_buffer::~cuda_buffer() { hipFree(_device_buffer); }

buffer_sptr cuda_buffer::make(size_t num_items,
                              size_t item_size,
                              std::shared_ptr<buffer_properties> buffer_properties)
{
    auto cbp = std::dynamic_pointer_cast<cuda_buffer_properties>(buffer_properties);
    if (cbp != nullptr) {
        return buffer_sptr(new cuda_buffer(num_items, item_size, cbp->buffer_type()));
    } else {
        throw std::runtime_error(
            "Failed to cast buffer properties to cuda_buffer_properties");
    }
}

int cuda_buffer::size()
{ // in number of items
    int w = _write_index;
    int r = _read_index;

    if (w < r)
        w += _buf_size;
    return (w - r) / _item_size;
}
int cuda_buffer::capacity() { return _num_items; }

void* cuda_buffer::read_ptr()
{
    if (_buffer_type == cuda_buffer_type::D2H) {
        return (void*)&_host_buffer[_read_index];
    } else {
        return (void*)&_device_buffer[_read_index];
    }
}
void* cuda_buffer::write_ptr()
{
    if (_buffer_type == cuda_buffer_type::H2D) {
        return (void*)&_host_buffer[_write_index];
    } else {
        return (void*)&_device_buffer[_write_index];
    }
}

bool cuda_buffer::read_info(buffer_info_t& info)
{
    // Need to lock the buffer to freeze the current state
    // if (!_buf_mutex.try_lock()) {
    //     return false;
    // }
    // _buf_mutex.lock();
    std::lock_guard<std::mutex> guard(_buf_mutex);

    info.ptr = read_ptr();
    info.n_items = size();
    info.item_size = _item_size;

    return true;
}

bool cuda_buffer::write_info(buffer_info_t& info)
{
    // if (!_buf_mutex.try_lock()) {
    //     return false;
    // }
    // _buf_mutex.lock();
    std::lock_guard<std::mutex> guard(_buf_mutex);

    info.ptr = write_ptr();
    info.n_items =
        capacity() - size() - 1; // always keep the write pointer 1 behind the read ptr
    if (info.n_items < 0)
        info.n_items = 0;
    info.item_size = _item_size;

    return true;
}

void cuda_buffer::post_read(int num_items)
{
    std::lock_guard<std::mutex> guard(_buf_mutex);
    // advance the read pointer
    _read_index += num_items * _item_size;
    if (_read_index >= _buf_size) {
        _read_index -= _buf_size;
    }
    // _buf_mutex.unlock();
}
void cuda_buffer::post_write(int num_items)
{
    std::lock_guard<std::mutex> guard(_buf_mutex);

    unsigned int bytes_written = num_items * _item_size;
    int wi1 = _write_index;
    int wi2 = _write_index + _buf_size;
    // num_items were written to the buffer
    // copy the data to the second half of the buffer

    int num_bytes_1 = std::min(_buf_size - wi1, bytes_written);
    int num_bytes_2 = bytes_written - num_bytes_1;

    if (_buffer_type == cuda_buffer_type::H2D) {
        hipMemcpy(&_device_buffer[wi1],
                   &_host_buffer[wi1],
                   bytes_written,
                   hipMemcpyHostToDevice);

        // memcpy(&_host_buffer[wi2], &_host_buffer[wi1], num_bytes_1);
        hipMemcpy(&_device_buffer[wi2],
                   &_device_buffer[wi1],
                   num_bytes_1,
                   hipMemcpyDeviceToDevice);
        if (num_bytes_2) {
            // memcpy(&_host_buffer[0], &_host_buffer[_buf_size], num_bytes_2);
            hipMemcpy(&_device_buffer[0],
                       &_device_buffer[_buf_size],
                       num_bytes_2,
                       hipMemcpyDeviceToDevice);
        }
    } else if (_buffer_type == cuda_buffer_type::D2H) {
        hipMemcpy(&_host_buffer[wi1],
                   &_device_buffer[wi1],
                   bytes_written,
                   hipMemcpyDeviceToHost);

        memcpy(&_host_buffer[wi2], &_host_buffer[wi1], num_bytes_1);
        // hipMemcpy(&_device_buffer[wi2],
        //            &_device_buffer[wi1],
        //            num_bytes_1,
        //            hipMemcpyDeviceToDevice);

        if (num_bytes_2) {
            memcpy(&_host_buffer[0], &_host_buffer[_buf_size], num_bytes_2);
            // hipMemcpy(&_device_buffer[0],
            //            &_device_buffer[_buf_size],
            //            num_bytes_2,
            //            hipMemcpyDeviceToDevice);
        }
    } else // D2D
    {
        hipMemcpy(&_device_buffer[wi2],
                   &_device_buffer[wi1],
                   num_bytes_1,
                   hipMemcpyDeviceToDevice);
        if (num_bytes_2)
            hipMemcpy(&_device_buffer[0],
                       &_device_buffer[_buf_size],
                       num_bytes_2,
                       hipMemcpyDeviceToDevice);
    }
    // advance the write pointer
    _write_index += bytes_written;
    if (_write_index >= _buf_size) {
        _write_index -= _buf_size;
    }

    // _buf_mutex.unlock();
}

void cuda_buffer::copy_items(std::shared_ptr<buffer> from, int nitems)
{
    std::lock_guard<std::mutex> guard(_buf_mutex);
    memcpy(write_ptr(), from->write_ptr(), nitems * _item_size);
}
} // namespace gr
