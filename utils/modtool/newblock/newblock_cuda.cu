/* -*- c++ -*- */
/*
 * Copyright <COPYRIGHT_YEAR> <COPYRIGHT_AUTHOR>
 *
 * This file is part of GNU Radio
 *
 * SPDX-License-Identifier: GPL-3.0-or-later
 *
 */

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>


// The block cuda file is just a wrapper for the kernels that will be launched in the work
// function
namespace gr {
namespace newmod {
__global__ void apply_newblock_kernel(const uint8_t* in, uint8_t* out, int batch_size)
{
    // block specific code goes here
}

void apply_newblock(
    const uint8_t* in, uint8_t* out, int grid_size, int block_size, hipStream_t stream)
{
    int batch_size = block_size * grid_size;
    apply_newblock_kernel<<<grid_size, block_size, 0, stream>>>(in, out, batch_size);
}

void get_block_and_grid(int* minGrid, int* minBlock)
{
    // https://developer.nvidia.com/blog/cuda-pro-tip-occupancy-api-simplifies-launch-configuration/
    hipOccupancyMaxPotentialBlockSize(minGrid, minBlock, apply_newblock_kernel, 0, 0);
}
} // namespace newmod
} // namespace gr