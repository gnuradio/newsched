#include <hip/hip_runtime.h>


__global__ 
void multiply_const_doublecopy_kernel(int n, float a, float* in, float* out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        out[i] = a * in[i];
}

namespace gr {
namespace cuda {
void multiply_const_doublecopy_kernel_wrapper(int N, float k, const float* in, float* out)
{
    float *dev_x, *dev_y;


    hipMalloc(&dev_x, N * sizeof(float));
    hipMalloc(&dev_y, N * sizeof(float));


    hipMemcpy(dev_x, in, N * sizeof(float), hipMemcpyHostToDevice);

    const int nthreads = 64;
    // Perform SAXPY on 1M elements
    multiply_const_doublecopy_kernel<<<(N + nthreads - 1) / nthreads, nthreads>>>(N, k, dev_x, dev_y);

    hipMemcpy(out, dev_y, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_x);
    hipFree(dev_y);
}

} // namespace cuda
} // namespace gr