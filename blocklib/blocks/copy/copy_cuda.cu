#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>


// The block cuda file is just a wrapper for the kernels that will be launched in the work
// function
namespace gr {
namespace blocks {
__global__ void apply_copy_kernel(const uint8_t* in, uint8_t* out, int batch_size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int n = batch_size;
    if (i < n) {
        out[i] = in[i];
    }
}

void apply_copy(
    const uint8_t* in, uint8_t* out, int grid_size, int block_size, hipStream_t stream)
{
    int batch_size = block_size * grid_size;
    apply_copy_kernel<<<grid_size, block_size, 0, stream>>>(in, out, batch_size);
}

void get_block_and_grid(int* minGrid, int* minBlock)
{
    // https://developer.nvidia.com/blog/cuda-pro-tip-occupancy-api-simplifies-launch-configuration/
    hipOccupancyMaxPotentialBlockSize(minGrid, minBlock, apply_copy_kernel, 0, 0);
}
} // namespace blocks
} // namespace gr